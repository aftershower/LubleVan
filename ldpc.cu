#include "hip/hip_runtime.h"

//LDPC decoder
//#define N 19942
//#define K 10003
//#define L 9939

#define N 8176
#define K 7156
#define L 1020

//#define N 9968
//#define K 4984
//#define L 4984

#define TOTAL_LENGTH 100
#define Eb_No 0.5
//#define FILENAME "PCM.5000.10000"

/*
#define N 2048
#define K 1024
#define L 1024
*/

#define ITERATION 200
#define MAX_Le 10.0
#define MIN_Le 0.0
#define _HH	printf("Have been Here!\n")
#define CLIP 12.0
#define min(x,y) ((x<=y) ? x:y)
#define max(x,y) ((x>=y) ? x:y)
#define sign(x) ((x <= 0) ? (-1.0):(1.0))
#define absd(x) ((x <= 0)? (-1.0*x):(x))
#define absi(x) ((x <= 0)? (-1*x):(x))

#include<stdio.h>
#include<math.h>
#include<memory.h>
#include<stdlib.h>


/****************************************************************************/
/*  RANDOM.H                                                                */
/*  Includes functions which simulate 
/*                                                                          */
/*    float ran2(long *idum) - generates random numbers uniformily         */
/*                              distributed over (0,1).  Random sequence is */
/*                              initialized by calling ran2 with *idum a    */
/*                              negative number.                            */
/*                                                                          */
/*    float gasdev(long *idum) - generates Gaussian random numbers, N(0,1) */
/*                              utilizing the function ran2.  Initialized   */
/*                              the same way as ran2.                       */
/*                                                                          */
/*    int irbit(unsigned long *iseed) - generates a pseudo-random noise     */
/*					See files "pn_gen#.h"		 */
/*    float poidev(float xm, long *idum) - generates an integer value     */
/*				that is a random deviate drawn from a 	    */
/* 				Poisson distribution of mean xm, using      */
/*				ran2(idum) as a source of the uniform dev.  */
/*									    */
/*    float gammln(float xx) - returns the value of the log of the gamma  */
/*                           	function evaluated at xx for xx>0           */
/*				(used for poidev()).			    */
/*    									    */
/*    float expdev(long *idum) - returns an exponentially distributed,     */
/* 				positive, random deviate of unit mean.      */
/*				To change mean to lambda, multiply the      */
/*				result of the call by lambda.		    */
/*                                                                          */
/*   Note:  requires that math.h be included !!!!!!!!!!!!!!!!!!!            */
/****************************************************************************/

/****************************************************************************/
/* function:  float ran2(long *idum)                                       */
/****************************************************************************/


#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define IM1     2147483563
#define IM2     2147483399
#define AM      (1.0/IM1)
#define IMM1    (IM1-1)
#define IA1     40014
#define IA2     40692
#define IQ1     53668
#define IQ2     52744
#define IR1     12211
#define IR2     3791
#define NTAB    32
#define NDIV    (1+IMM1/NTAB)
#define EPS     1.2e-7
#define RNMX    (1.0-EPS)
#define PI	3.1415927

float ran2(long *idum)                 /////////////////产生均匀分布的随机数。。
{
	int j;
	long k;
	static long idum2=123456789;
	static long iy=0;
	static long iv[NTAB];
	float temp;

	if (*idum <= 0)
	{
		if (-(*idum) < 1)
			*idum = 1;
		else
			*idum = -(*idum);
		idum2 = (*idum);
		for (j=NTAB+7 ; j>=0 ; j--)
		{
			k = (*idum)/IQ1;
			*idum = IA1*(*idum-k*IQ1)-k*IR1;
			if (*idum < 0)
				*idum +=IM1;
			if (j < NTAB)
				iv[j] = *idum;
		}
		iy = iv[0];
	}
	k = (*idum)/IQ2;                
	*idum = IA1*(*idum-k*IQ1)-k*IR1;
	if (*idum < 0)
		*idum += IM1;
	k = idum2/IQ2;
	idum2=IA2*(idum2-k*IQ2)-k*IR2;
	if (idum2 < 0)
		idum2 += IM2;
	j = iy/NDIV;
	iy = iv[j]-idum2;
	iv[j] = *idum;
	if (iy < 1)
		iy += IMM1;
	if ( (temp=AM*iy) > RNMX)
		return RNMX;
	else
		return temp;
}


/****************************************************************************/
/* function:  float gasdev(long *idum)                                     */
/****************************************************************************/

float gasdev(long *idum)               ///////////高斯噪声 
{
	float ran2(long *idum);
	static int iset = 0;
	static float gset;
	float fac, rsq, v1, v2;

	if (iset == 0)
	{
		do {
			v1 = 2.0*ran2(idum)-1.0;
			v2 = 2.0*ran2(idum)-1.0;
			rsq = v1*v1 + v2*v2;
		} while (rsq >= 1 || rsq == 0.0);
		fac = sqrt(-2.0*log(rsq)/rsq);
		gset = v1*fac;
		iset = 1;
		return v2*fac;
	}
	else {
		iset = 0;
		return gset;
	}
}

/*****************************************************************************/
/* function:  float poidev(float xm, long *idum)                           */
/*****************************************************************************/

float poidev(float xm, long *idum)            ///////////////柏松分布。。。

{
	float gammln(float xx);
	float ran2(long *idum);
	static float sq, alxm, g, oldm=(-1.0);
	float em, t, y;

	if (xm < 12.0) {
		if (xm != oldm) {
			oldm=xm;
			g=exp(-xm);
		}
		em = -1;
		t=1.0;
		do {
			++em;
			t *= ran2(idum);
		} while (t>g);
	} else {
		if (xm != oldm) {
			oldm=xm;
			sq=sqrt(2.0*xm);
			alxm=log(xm);
			g=xm*alxm-gammln(xm+1.0);
		}
		do {
			do {
				y=tan(PI*ran2(idum));
				em=sq*y+xm;
			} while (em < 0.0);
			em=floor(em);
			t=0.9*(1.0+y*y)*exp(em*alxm-gammln(em+1.0)-g);
		} while (ran2(idum) > t);
	}
	return em;
}


/*****************************************************************************/
/* function:  float gammln(float xx)                                       */
/*****************************************************************************/

float gammln(float xx)                 			//////////计算对数值？？？？？？？？？           
{
	float x,y,tmp,ser;
	static float cof[6]={76.18009172947146, -86.50532032941677,
		24.01409824083091, -1.231739572450155, 0.1208650973866179e-2,
		-0.5395239384953e-5};
	int j;

	y=x=xx;
	tmp=x+5.5;
	tmp -= (x+0.5)*log(tmp);
	ser=1.000000000190015;
	for (j=0; j<=5; j++) 
		ser += cof[j]/++y;
	return -tmp+log(2.5066282746310005*ser/x);
}

/*****************************************************************************
 * function:  float expdev(long *idum)					     *
 *****************************************************************************/
 
 float expdev(long *idum)                   //////////////指数分布？/////////
 {
 	float dum;
 	
 	do
 		dum=ran2(idum);
 	while (dum==0.0);
 	return -log(dum);
 }


typedef struct RECORD1
{
int no_elem;
int *pos;
int *pos2;
int count;
} record1;

record1 row[L],col[N];
float stdd,probzero,nvi;



int parity_check(unsigned short c[])        ////////////////////检验
//unsigned short c[];
{
int i,j,k,sum;
for(i=0;i<=L-1;i++)
	{
	sum = 0;
	for (j=0;j<=row[i].no_elem-1;j++)
		{
		sum = (sum + c[row[i].pos[j]])%2;
		}
	if (sum != 0)
		return(0);
	}
return(1);
}
		
float boxminus(float x,float y)
{
float z;

z = log((exp(x)-exp(y))/(1-exp(x+y)));                 ///////////////某个函数。。

return(z);
}

float psifunc(float x)
{
return(log(tanh(absd(x/2))));             /////////////双曲正切函数的对数。。。（对数似然用？）
}

		
float boxplus( float x,float  y)
{
float z;

z = log((exp(x)+exp(y))/(1+exp(x+y)));           ///////////某个函数  类似于boxminus（x，y）。。。。


return(z);
}

void main()                       //////////////main函数
{
int i,j,k,l,np,m,n,p,q,simpt;
unsigned short d[K],c[N],dec[N];
int count,N_ITER,iter,itemp[100];               ////////////////////n=1000
int cf,pkt_err,detner,undetner,nber;
float detber,undetber,totber,fer;
long seed;
float ccia[]={Eb_No};
float cci,cdr,const1,**Lc,**Le,Lch[N],Lf[N],temp,r[N],tmp;
float tmpmag1,tmpsign1,tmpmag2,tmpsign2,tmparr[N];
FILE *Gmat,*Hmat,*fpres,*fptemp1,*fptemp2;
char tstr[80],tc='0';
int checkH[L];
FILE *fp;
int nozero;
char FILENAME[100];

for (i=0;i<L;i++) checkH[i]=0;	

         /////////////////fopen

sprintf(FILENAME,"PCM.%d.%d", K,N);           //////////////sprintf返回值：字符串长度
Hmat = fopen("D:\\Code\\C\\asdfsadfg\\asdfsadfg\\PCM.8176.7156","r");

fgets(tstr,80,Hmat);                                //////////////fgets从流读取n-1字符除非读完行参数s来接收字符串成功则返回tstr指针否则返回NULL
for(i=0;i<=N-1;i++)
        {
        fscanf(Hmat,"<%d> ",&l);                                //////////////fscanf返回值：返回实际被转换并赋值的输入项的数目。返回成功读入的个数
        col[i].no_elem = l;
        col[i].pos = (int *) calloc(col[i].no_elem,sizeof(int)); /////calloc   分配l个sizeof单元  并返回地址
        for (j=0;j<=col[i].no_elem-1;j++)
                {
                fscanf(Hmat,"%d  ",&l); 
                col[i].pos[j] = l;
//		fprintf(fp,"[%d] ",col[i].pos[j]);
				checkH[l]=1;
                (row[l].no_elem)++;
                }
        fscanf(Hmat," \n",tc);
        }

nozero=1;
for (i=0;i<L;i++) 
{ if (checkH[i]==0)
	{ printf("i=%d \n", i);
		nozero=0;
		}	
}
if (nozero) printf("nozero!!!!\n");
printf("Successfully read H matrix \n");
fflush(stdout);           //////////////////////fflush(stdout)刷新标准输出缓冲区把输出缓冲区里东西打印标准输出设备上
/*///////////////////////////////////////////////////////////////////
//////////////////////////////////////////////
//////////////////////////////////////////////////////////*/
for(i=0;i<=L-1;i++)
        {
        row[i].pos = (int *) calloc(row[i].no_elem,sizeof(int));
        row[i].pos2 = (int *) calloc(row[i].no_elem,sizeof(int));
        row[i].count = 0;
        }

for(i=0;i<=N-1;i++)
        {
        for(j=0;j<=col[i].no_elem-1;j++)
                {
                k = col[i].pos[j];
                row[k].pos[row[k].count] = i;
                row[k].pos2[row[k].count] = j;
                /*if (row[k].count >= row[k].no_elem)
                        printf("something wrong row index exceeds \n");*/
                (row[k].count)++;
                }
	}
	/*    可能是在产生矩阵///////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////*/
printf("successfully formed row and col positions \n");

Le = (float **)calloc(N,sizeof(float *));
Lc = (float **)calloc(N,sizeof(float *));
for(i=0;i<=N-1;i++)
	{
	Le[i] = (float *)calloc(col[i].no_elem,sizeof(float));
	Lc[i] = (float *)calloc(col[i].no_elem,sizeof(float));                                          ///////////////？？？
	if ((Le[i] == NULL)||(Lc[i] == NULL))
		 printf("problem in allocated memory for %d \n",i);
		 system("pause");
	}


seed = -108; N_ITER = ITERATION;

for(simpt=0;simpt<=0;simpt++)
	{
	cci = ccia[simpt]-10.0*log10((float)(N)/(float)(K));
	stdd = sqrt(0.5/(pow(10.0,(float)cci/10.0))); 
	
	printf("std = %f \n",stdd);
	nvi = 1.0/(2.0*stdd*stdd);const1 = -2.0/(stdd*stdd);               //////////////const1
	probzero = exp(-1.0*nvi);
	

	pkt_err = 0;undetner=0;detner=0;
	
	for(np=1;((np<=TOTAL_LENGTH) && (pkt_err <= 50));np++)
		{
		for(i=0;i<=K-1;i++)
			{
			/*d[i] = (unsigned short)(ran2(&seed)*2.0);*/
			d[i] = 0;
			}
		
		/*encode(d,c,G);*/

		for(i=0;i<=N-1;i++)
			c[i]=0;
	

		if (parity_check(c) != 1) 
			printf("Something wrong parity check is not satisfied \n"); 
		else
			printf("Parity check satisfied \n");
		
		for (i=0;i<=N-1;i++)
			{
			r[i] = (float)(2*c[i]-1)+gasdev(&seed)*stdd;            ///////////////加噪声
			}
	
		for (i=0;i<=N-1;i++)
			{
			Lch[i] = const1*r[i];
			for(j=0;j<=col[i].no_elem-1;j++)
				{
				Lc[i][j] = Lch[i];
				}
			}

		cf = 0;
		printf("Eb/No = %f packet %d iter = ",ccia[simpt],np);
		_HH;
		for (iter=1;((iter<=N_ITER) && (cf != 1));iter++)              //////////////迭代译码？
			{
//			printf("[%d] ",iter);
//_HH;
			for (j=0;j<=L-1;j++)															/////for1main
				{
				
				for(l=0;l<=row[j].no_elem-1;l++)									//for1of1					
					{
					q = row[j].pos2[l];
					p = row[j].pos[l];
					tmparr[l] = psifunc(Lc[p][q]);              /////////////////双曲正切函数的对数
					}
				tmpmag1 = tmparr[0]; 

				tmpsign1 = sign(Lc[row[j].pos[0]][row[j].pos2[0]]);
				
				for(l=1;l<=row[j].no_elem-1;l++)								///for2of1
                                	{
                                        p = row[j].pos[l];
					q = row[j].pos2[l];
					tmpmag1 = tmpmag1 + tmparr[l];
					tmpsign1 = tmpsign1*sign(Lc[p][q]);
					if (tmpsign1<-CLIP) tmpsign1=-CLIP;
					if (tmpsign1>CLIP) tmpsign1=CLIP;
                                        }
			//	_HH;
				for(i=0;i<=row[j].no_elem-1;i++)							/////for3of1
					{
                                        p = row[j].pos[i];
					q = row[j].pos2[i];
					tmpmag2 = tmpmag1 - tmparr[i];
					tmpsign2 = tmpsign1*sign(Lc[p][q]);

					Le[p][q] = tmpsign2*(-psifunc(tmpmag2));
					if (Le[p][q]<-CLIP) Le[p][q]=-CLIP;
					if (Le[p][q]>CLIP) Le[p][q]=CLIP;
					}
//					_HH;
				}				
				
			for(i=0;i<=N-1;i++)                          ////for cycle1
				{
				Lf[i] = Lch[i];
				for(j=0;j<=col[i].no_elem-1;j++)
                                        {
					Lf[i] += Le[i][j];										////checkNode1  lf=N   Le=N*no_elem
					}
				for(j=0;j<=col[i].no_elem-1;j++)
                                        {
					Lc[i][j] = Lf[i] - Le[i][j];											////checkNode2
					}
				}

			for (i=0;i<=N-1;i++)                   //////////////for2
				{
				if (Lf[i] > 0.0) dec[i] = 0;
					else dec[i] = 1;
				Lf[i] = Lf[i]-Lch[i];
				}
//		printf("Lf[i]=%e\n",Lf[229]);
                nber = 0;
            for (i=L;i<=N-1;i++) 						/////////////for3
                        {
                        if (dec[i] != d[i-L]) nber++;
                        }

//		printf("After %d iterarions there are %d errors \n",iter,nber);
				
			cf = parity_check(dec);
			if (cf == 1) printf("\n parity check satisfied during %d iter can stop iterations \n",iter);
			}/* for iter = */

		if (cf != 1) pkt_err++;
		nber = 0;
		for (i=L;i<=N-1;i++)
			{
			if (dec[i] != d[i-L]) nber++;
			}
		if (cf != 1) detner = detner + nber;
		else
		undetner = undetner + nber; 

		undetber = (float)undetner/(float)((np)*K);
		detber = (float)detner/(float)((np)*K);
		totber = (float)undetber + detber;
		fer = (float)pkt_err/(float)(np);

		printf("Eb/No = %f K = %d N = %d L = %d \n",ccia[simpt],K,N,L);
		printf("Detected BER = %e Undetected BER = %e Total BER = %e ",detber,undetber,totber);
		printf("FER = %e \n \n",fer);
		}/* for np = */

		undetber = (float)undetner/(float)((np-1)*K);
		detber = (float)detner/(float)((np-1)*K);
		totber = (float)undetber + detber;
		fer = (float)pkt_err/(float)(np-1);

		fpres =	fopen("RESULTS_91","a");
		fprintf(fpres,"Eb/No = %f K = %d N = %d L = %d \n",ccia[simpt],K,N,L);
		fprintf(fpres,"Detected BER = %e Undetected BER = %e Total BER = %e ",detber,undetber,totber);
		fprintf(fpres,"FER = %e \n \n",fer);
		fclose(fpres);

	} /* for simpt */ 
} /* for main */
